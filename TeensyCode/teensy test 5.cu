#include "hip/hip_runtime.h"
// Three bytes interface is used between Arduino Due and Teensy on COM serial port:
// Byte 1: 255 (sync byte)
// Byte 2: <control> - [0:255]
// Byte 3: <value MSB> - [0:255]

// --------------------------------------------------
#include <MCP492X.h> // Include the library
#include <SPI.h>
#include <lfo.h>          // required for function generation
#include <adsr.h>         // required for function generation

#define CHIP_SELECT_A 4 // =SC1 on schematic 
//#define CHIP_SELECT_B 3 // =SC2 on schematic etc
#define CHIP_SELECT_C 10
#define CHIP_SELECT_D 9
#define CHIP_SELECT_E 8
#define CHIP_SELECT_F 7 
#define CHIP_SELECT_G 6
#define CHIP_SELECT_H 5


// DAC init: DAC_A holds DAC1 and 2, DAC_B holds DAC3 and 4 and so on
MCP492X myDacA(CHIP_SELECT_A);
//MCP492X myDacB(CHIP_SELECT_B);
MCP492X myDacC(CHIP_SELECT_C);
MCP492X myDacD(CHIP_SELECT_D);
MCP492X myDacE(CHIP_SELECT_E);
MCP492X myDacF(CHIP_SELECT_F);
MCP492X myDacG(CHIP_SELECT_G);
MCP492X myDacH(CHIP_SELECT_H);

#define DACSIZE 4096             // vertical resolution of the DACs

// NOTE: all ID_xxxx variables below are CONTROL numbers for interfacing with Arduino Due
//       for instance: if Arduino sends over 255 - 210 - 0 - 100 that will mean that the attack of the ADSR is going to be set to 100µs
//       (Because of this definition -> #define ID_ADSR_ATTACK 210)
//NOTE2: you can change these definitions as you like, but dont forget to change the according numbers in the DUE code as well.

#define ID_SONG_BPM 250       // defines the BPM
#define ID_NOTE_PRIORITY 251

#define ID_VOLUME 201
#define ID_VCO1_OCT 202
#define ID_VCO1_LVL 203

#define ID_CUTOFF_POSITION 206
#define ID_VCF_RES 213


// for ADSR1
#define ID_ADSR1_ATTACK 10
#define ID_ADSR1_DECAY 11
#define ID_ADSR1_SUSTAIN 12
#define ID_ADSR1_RELEASE 13

// for LFO1
#define ID_LFO1_MODE 100
#define ID_LFO1_MODE0_FREQ 101
#define ID_LFO1_MODE1_RATE 102
#define ID_LFO1_WAVEFORM 103
#define ID_LFO1_AMPL 104
#define ID_LFO1_AMPL_OFFSET 105
#define ID_LFO1_PHASE 106

//Destinations

#define ID_PB1_ADSR1_AMOUNT 30
#define ID_PB1_LFO1_AMOUNT 130

#define ID_SHAPE1_ADSR1_AMOUNT 38
#define ID_SHAPE1_LFO1_AMOUNT 138

#define ID_LVL1_ADSR1_AMOUNT 42
#define ID_LVL1_LFO1_AMOUNT 142

#define ID_CUTOFF_ADSR1_AMOUNT 66
#define ID_CUTOFF_ADSR2_AMOUNT 67
#define ID_CUTOFF_LFO1_AMOUNT 166
#define ID_CUTOFF_LFO2_AMOUNT 167

#define ID_RES_ADSR1_AMOUNT 70
#define ID_RES_LFO1_AMOUNT 170




// internal variables
int rx_state = 0;
int i=0;
byte cc_sync;
byte cc_control;
byte cc_val1;
byte cc_val2;

uint8_t pitchBendChan;

unsigned long   t = 0;
unsigned long   sync_t0 = 0;
unsigned long   connected_t0 = 0;


float _freqArray[24] = {64, 48, 32, 24, 16, 12, 8, 6, 5.3333, 4, 3.2, 3, 2.667, 2, 1.333, 1, 0.667, 0.5, 0.333, 0.25, 0.167, 0.125, 0.0625, 0.03125}; //array representing the different
                                                                                                                                                      //bpm subdivision for sync lfos
float           bpm = 120;

int             note_priority = 2; // 0 = bottom note 1=top note 2=last note //receive from teensy in note priority menu default is last note 

int             VCO1_OCT = 0;
int             VCO2_OCT =0;
int             adsr1_sample = 0; //current sample from modulation to be multiplied by amounts to relevant DACs
int             lfo1_sample = 0;
int             lfo1_ampl= DACSIZE-1;
int             VCO1_SHAPE=0;
int             VCO2_SHAPE=0;

// internal classes
lfo         lfo1(DACSIZE);
adsr        adsr1(DACSIZE); 

//ints for writing to DACS
int pb;
int mV;
int             PB1=0;  
int             PB2=0;
int             CUTOFF=0;
int             RES=0;
int             VCA1=0;
int             VCA2=0;
int             VCA3=0;
int             VCA4=0;
int             VCA5=0;
int             VCA6=0;
int             VCA7=0;
int             VCA8=0;

//parameter values, range 0/1
int           lvl1_position = 0;

int           cutoff_position = 0;
int           res_position = 1;

int           vca4_position = 0.;
int           vca5_position = 0.;

float          volume_position = 0.4; 

//modulation amounts value, range -1/1
float           PB1_ADSR1_AMOUNT = 0.;
float           PB1_LFO1_AMOUNT = 0.;

float           PB2_ADSR1_AMOUNT = 0.;
float           PB2_LFO1_AMOUNT = 0.;

float           LVL1_ADSR1_AMOUNT = 0.;
float           LVL1_LFO1_AMOUNT = 0.;

float           LVL2_ADSR1_AMOUNT = 0.;
float           LVL2_LFO1_AMOUNT = 0.;

float           CUTOFF_ADSR1_AMOUNT = 0.;
float           CUTOFF_LFO1_AMOUNT = 0.;

float           RES_ADSR1_AMOUNT = 0.;
float           RES_LFO1_AMOUNT = 0.;

float           DRIVE_ADSR1_AMOUNT = 0.;
float           DRIVE_LFO1_AMOUNT = 0.;

float           SHAPE1_ADSR1_AMOUNT = 0.;
float           SHAPE1_LFO1_AMOUNT = 0.;

float           SHAPE2_ADSR1_AMOUNT = 0.;
float           SHAPE2_LFO1_AMOUNT = 0.;

float           VCA7_ADSR1_AMOUNT = 0.;
float           VCA7_LFO1_AMOUNT = 0.;

void setup() {
  delay(100);
  lfo1.setAmpl(DACSIZE); // init amplitude

  sync_t0 = t;
  connected_t0 = t;

 // pinMode(LED_BUILTIN, OUTPUT);
 // digitalWrite(LED_BUILTIN, false);
  
  myDacA.begin();
//  myDacB.begin();
  myDacC.begin();
  myDacD.begin();
  myDacE.begin();
  myDacF.begin();
  myDacG.begin();
  myDacH.begin();
  lfo1.setMode(0);
  lfo1.setWaveForm(2); //triangle wave
  lfo1.setMode0Freq(1);

  Serial.begin(9600);

}

unsigned long trigTimer[3] = {0}; //from elkayem code, don't kno why it's in between setup and main and not in init stage
bool notes[3][88] = {0}, initial_loop = 1; 
int8_t noteOrder[3][10] = {0}, orderIndx[3] = {0};


void loop() {

    t = micros();           // take timestamp
    int8_t noteMsg, velocity, channel, d2;

    if (usbMIDI.read()) {                    
    byte type = usbMIDI.getType();
   // Serial.print(type);
    if (type == 224){
        // Pitch bend output from 0 to 1023 mV.  Left shift d2 by 4 to scale from 0 to 2047.
        // With DAC gain = 1X, this will yield a range from 0 to 1023 mV.  Additional amplification
        // after DAC will rescale to -1 to +1V.
        d2 = usbMIDI.getData2(); // d2 from 0 to 127, mid point = 64
        pb= d2<<4;  
        //Serial.print(pb);
        myDacA.analogWrite(1,pb);
        }
        
   
    switch (type) {
    case usbMIDI.NoteOff:
    case usbMIDI.NoteOn:
        noteMsg = usbMIDI.getData1() - 21; // A0 = 21, Top Note = 108
        channel = usbMIDI.getChannel()-1;
        
        if (channel > 2) return;  // Only channel 0,1,2 supported
        if ((noteMsg < 0) || (noteMsg > 87)) break;  // Only 88 notes of keyboard are supported
        if (type == usbMIDI.NoteOn) velocity = usbMIDI.getData2();
        else velocity  = 0;  
        if (velocity == 0)  {
        notes[channel][noteMsg] = false;
        adsr1.noteOff(t);
        }
        else {
        notes[channel][noteMsg] = true;
        adsr1.noteOn(t);
        }
        if (note_priority == 1){ // Top note priority
          mV=commandTopNote(channel);
          myDacA.analogWrite(0,mV);

        }
        else if (note_priority == 0){ // Bottom note priority
          mV= commandBottomNote(channel);
          myDacA.analogWrite(0,mV);
        }
        else { // Last note priority  
          if (notes[channel][noteMsg]) {  // If note is on and using last note priority, add to ordered list
              orderIndx[channel] = (orderIndx[channel]+1) % 10;
              noteOrder[channel][orderIndx[channel]] = noteMsg;                 
          }
         mV = commandLastNote(channel);
         myDacA.analogWrite(0,mV);
        }
            
        break;
       // case usbMIDI.PitchBend:
       // if (usbMIDI.getChannel() == pitchBendChan) 
         
        //break;
    }
}

    adsr1_sample = adsr1.getWave(t); //current sample from modulation (0 centered) to be multiplied by amounts to relevant DACs
    lfo1_sample = lfo1.getWave(t);

    //-------------------------------write to CUTOFF---------------------------------//
    CUTOFF=int(cutoff_position+ adsr1_sample*CUTOFF_ADSR1_AMOUNT + lfo1_sample*CUTOFF_LFO1_AMOUNT ); //all modulations weighted and added together 
    if (CUTOFF > 4095){
        CUTOFF=4095;
    }
        if (CUTOFF < 0){
        CUTOFF=0;
    }
    myDacD.analogWrite(1,CUTOFF); 

    //-------------------------------write to RES---------------------------------//
    RES=int(res_position + adsr1_sample*RES_ADSR1_AMOUNT + lfo1_sample*RES_LFO1_AMOUNT ); //all modulations weighted and added together 
    if (RES > 4095){
        RES=4095;
    }
        if (RES < 0){
        RES=0;
    }
    myDacD.analogWrite(0,RES); 


     //-------------------------------write to wavemixer vca4---------------------------------//
    VCA4=int(cos(SHAPE1)*(lvl1_position + adsr1_sample*LVL1_ADSR1_AMOUNT + lfo1_sample*LVL1_LFO1_AMOUNT )); 

    if (VCA4 > 4095){
        VCA4=4095;
    }
    if (VCA4 < 0 ){
        VCA4=0;
    }
    myDacF.analogWrite(1,VCA4);
     
     
     //-------------------------------write to wavemixer vca5---------------------------------//
    VCA5=int(sin(SHAPE1)*(lvl1_position + adsr1_sample*LVL1_ADSR1_AMOUNT + lfo1_sample*LVL1_LFO1_AMOUNT )); 
    if (VCA5 > 4095){
        VCA5=4095;
    }
    if (VCA5 < 0 ){
        VCA5=0;
    }
    myDacG.analogWrite(0,VCA5); 

    //-------------------------------write to OUTPUT VCA---------------------------------//
    VCA8=int(volume_position*(adsr1_sample)*(velocity/127)); //hard coded adsr1 to control volume enveloppe at all time, no volume mod coded for this iteration
    if (VCA8 > 4095){
        VCA8=4095;
    }
    if (VCA8 < 0 ){
        VCA8=0;
    }
    myDacH.analogWrite(1,VCA8);    




//----------Check if control commands have been received from Arduino------------//
  if (Serial.available()) {
    connected_t0 = t;
    if (digitalRead(LED_BUILTIN) == 0)
      digitalWrite(LED_BUILTIN, 1);
      
    rx_state++;
    switch (rx_state) {
      case 1:                     // first byte is always 255 for sync
        cc_sync = Serial.read();
        if(cc_sync != 255) {     // reset if first is not 255 sync byte
          rx_state = 0;
        }
        break;
      case 2:                     // second is the control byte / ID byte
        cc_control = Serial.read();
        break;        
      case 3:                     // third is the most significant byte of the value
        cc_val1 = Serial.read();     
        break;
      case 4:                     // fourth is the least significant byte of the value
        cc_val2 = Serial.read();
        rx_state = 0;

        // re-compile value from its two bytes (cc_val1 is the MSB and cc_val2 the LSB)
        int value = getInt(cc_val1, cc_val2);

        // Track specific IDs
        if (cc_control == ID_SONG_BPM) {
            bpm = ((float)value)/10;
            lfo1.setMode1Bpm(bpm);

        }
        if (cc_control == ID_NOTE_PRIORITY){
            note_priority = value; // 0 1 2
        }
              
         // LFO1
        else if (cc_control == ID_LFO1_MODE){ // 0 free running 1 bpm synced (button control)
            lfo1.setMode(value);
        }
        else if (cc_control == ID_LFO1_MODE0_FREQ){ //in Hz 
            lfo1.setMode0Freq(float(pow(10,value/5)), t);
        }
        else if (cc_control == ID_LFO1_MODE1_RATE){
            if (value > 96){
                value = 96;
            }
            lfo1.setMode1Rate(_freqArray[(int)value/5]);
        }
        else if(cc_control == ID_LFO1_AMPL) {
            lfo1_ampl = int(value/100*(DACSIZE-1)); // 0 to dacsize-1
            lfo1.setAmpl(lfo1_ampl);
        }
        else if(cc_control == ID_LFO1_AMPL_OFFSET){
            lfo1.setAmplOffset(int(value/100*(DACSIZE-1)));
        }
        else if(cc_control == ID_LFO1_WAVEFORM){
            lfo1.setWaveForm(int(value/25)); // range 0 to 4 
        }
        else if(cc_control == ID_LFO1_PHASE){
            lfo1.setMode1Phase((int(value*0.36)); //0 to 360 deg
        }

        // ADSR1
        else if (cc_control == ID_ADSR1_ATTACK)
            adsr1.setAttack(int(value/100*DACSIZE));                                          // times 1000 -> conversion from ms to µs  
        else if (cc_control == ID_ADSR1_DECAY)
            adsr1.setDecay(int(value/100*DACSIZE));                                           // times 1000 -> conversion from ms to µs  
        else if (cc_control == ID_ADSR1_SUSTAIN)
            adsr1.setSustain(pow(10, -7*(float)value/100)*(DACSIZE - 1));        // parameter is logarithmic from 0 to -70dB -> in Arduino range 0/100 for transportation to the teensy. To convert to 0/-70dB -> time -7 and divide by 10. Then we need to convet from log to lin, which is done with 10^x/10 -> therefore we divide by 1000 here.
        else if (cc_control == ID_ADSR1_RELEASE)
            adsr1.setRelease(1000*value);                                         // times 1000 -> conversion from ms to µs  

        // modulation amounts to pitch bend 1
        else if (cc_control == ID_PB1_ADSR1_AMOUNT){
            PB1_ADSR1_AMOUNT = float(value-50)/100) ;
        }
        else if (cc_control == ID_PB1_LFO1_AMOUNT){
            PB1_LFO1_AMOUNT = float(value-50)/100) ;
        }

        // modulation amounts to vco1 shape
        else if (cc_control == ID_SHAPE1_ADSR1_AMOUNT){
            SHAPE1_ADSR1_AMOUNT = float(value-50)/100) ;
        }
        else if (cc_control == ID_SHAPE1_LFO1_AMOUNT){
            SHAPE1_LFO1_AMOUNT = float(value-50)/100) ;
        }

        // modulation amounts to vco1 volume
        else if (cc_control == ID_LVL1_ADSR1_AMOUNT){
            LVL1_ADSR1_AMOUNT  = float(value-50)/100) ;
        }
        else if (cc_control == ID_LVL1_LFO1_AMOUNT){
            LVL1_LFO1_AMOUNT = float(value-50)/100) ;
        }

        // modulation amounts to vcf cutoff
        else if (cc_control == ID_CUTOFF_ADSR1_AMOUNT){
            CUTOFF_ADSR1_AMOUNT  = float(value-50)/100) ;
        }
        else if (cc_control == ID_CUTOFF_LFO1_AMOUNT){
            CUTOFF_LFO1_AMOUNT = float(value-50)/100) ;
        }

        // modulation amounts to vcf RESONANCE
        else if (cc_control == ID_RES_ADSR1_AMOUNT){
            RES_ADSR1_AMOUNT  = float ((value-50)/100) ;
        }
        else if (cc_control == ID_RES_LFO1_AMOUNT){
            RES_LFO1_AMOUNT = float((value-50)/100) ;
        }

        // parameter positions 
        else if (cc_control == ID_VCO1_OCT){ //still to add effect in mV value
            VCO1_OCT=value -2 ;
        }

        else if (cc_control == ID_VCO1_LVL){
            lvl1_position = int((value/100)*DACSIZE);
        }

        else if (cc_control == ID_CUTOFF_POSITION ){
            cutoff_position = int(value/100)*DACSIZE ;
        }
        else if (cc_control == ID_VCF_RES){
            res_position = int(value/100)*DACSIZE ;
        }
        else if (cc_control == ID_VOLUME){
            volume_position = float(value/100) ;
        }
        

        break;
    }
  }
}

int getInt(int l_highByte, int l_lowByte) {
  return ((unsigned int)l_highByte << 8) + l_lowByte;
}

int commandTopNote(int channel){
  int topNote = 0;
  bool noteActive = false;
 
  for (int i=0; i<88; i++){
    if (notes[channel][i]) {
      topNote = i;
      noteActive = true;
    }
  }

  if (noteActive){
    return note2mV(topNote);
      }
  else{ // All notes are off, turn off gate
  // turn adsrs off
   adsr1.noteOff(t);

  }
}
int commandBottomNote(int channel){

  int bottomNote = 0;
  bool noteActive = false;
 
  for (int i=87; i>=0; i--){
    if (notes[channel][i]) {
      bottomNote = i;
      noteActive = true;
    }
  }

  if (noteActive){
    return note2mV(bottomNote);
   // myDacA.analogWrite(0,note2mV(bottomNote));
  }
  else{ // All notes are off, turn off gate
   // turn adsrs off
   adsr1.noteOff(t);

  }
}
int commandLastNote(int channel){

  int8_t noteIndx;
  
  for (int i=0; i<10; i++) {
    noteIndx = noteOrder[channel][ mod(orderIndx[channel]-i, 10) ];
    if (notes[channel][noteIndx]) {

    //  myDacA.analogWrite(0,note2mV(noteIndx));
      
      return note2mV(noteIndx);
    }
  }
  // turn adsrs off
  adsr1.noteOff(t);

}

// Rescale 88 notes to 4096 mV:
//    noteMsg = 0 -> 0 mV 
//    noteMsg = 87 -> 4096 mV
// DAC output will be (4095/87) = 47.069 mV per note, and 564.9655 mV per octave
// Note that DAC output will need to be amplified by 1.77X for the standard 1V/octave 
#define NOTE_SF 47.069f 

int note2mV(int noteMsg) {
  //turn ADSRs ON
  adsr1.noteOn();
  unsigned int mV = (unsigned int) ((float) noteMsg * NOTE_SF * 1. + 0.5);
//  Serial.print("inside func");
//  Serial.print(mV);
  return mV;   
 
}

int mod(int a, int b){
    int r = a % b;
    return r < 0 ? r + b : r;
}